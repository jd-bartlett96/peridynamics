#include "hip/hip_runtime.h"
#include <math.h>
#include <inttypes.h>


__device__ __constant__ float L = Lr;
__device__ __constant__ int NX = NXr;
__device__ __constant__ int NY = NYr;
__device__ __constant__ int NZ = NZr;
__device__ __constant__ int NB = NBr;
__device__ __constant__ int NN = NNr;
__device__ __constant__ double L0s[];
__device__ __constant__ int jadd[];
__device__ __constant__ double dt = dtr;
__device__ __constant__ double ntau = ntaur;
__device__ __constant__ double rho = rhor;
__device__ __constant__ double ecrit = ecritr;

__device__ __constant__ double dlmlt = dlmltr;
__device__ __constant__ double fmlt = fmltr;
__device__ __constant__ double kappa = kappar;
__device__ __constant__ double am = amr;
__device__ __constant__ double mvec = mvecr;

__device__ __constant__ double hrad = hradr;
__device__ __constant__ double alpha = alphar;

__device__ __constant__ float xh = xhr;
__device__ __constant__ float xl = xlr;
__device__ __constant__ float yh = yhr;
__device__ __constant__ float yl = ylr;
__device__ __constant__ float zh = zhr;
__device__ __constant__ float zl = zlr;

__device__ __constant__ int penal = penalr;

__device__ __constant__ float HLF = 0.5;
__device__ __constant__ float TRE = 3;

__device__ __constant__ double ZER = 0;
__device__ __constant__ double ONE = 1;
__device__ __constant__ double TWO = 2;

__device__ int tt = 0;

__device__ __shared__ float sh_Sf[SHr]; // 4*NB + 1

__device__ bool Chi(float x, float y, float z){
    return (x>=xl) && (x<=xh) && (y>=yl) && (y<=yh) && (z>=zl) && (z<=zh);
}

__device__ bool TestBit(bool A[],  int64_t k ){
    return ( (A[k/8] & (1 << (k%8) )) != 0 ) ;     
}

__device__ void  SetBit(bool A[],  int64_t k ){
    A[k/8] |= 1 << (k%8);
}

__global__ void calcVolume(float *d_Sf, double *d_m, bool *d_dmg){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF) + xl;
    float yi = L*(j+HLF) + yl;
    float zi = L*(k+HLF) + zl;

    if (Chi(xi,yi,zi)) {
        double mi = ZER;
        for (int64_t b = 0;b<NB;b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                double L0 = L0s[b];
                mi += L0*L*L*L;
            }
        }
        d_m[iind] = mi;
    }
}

__global__ void calcDilation(float *d_Sf, double *d_u, double *d_dil, bool *d_dmg, double *d_W, double *d_m){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(iind==0){
        tt += 1;
    }

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF) + xl;
    float yi = L*(j+HLF) + yl;
    float zi = L*(k+HLF) + zl;

    if (Chi(xi,yi,zi)) {
        double ui = d_u[iind];
        double vi = d_u[NN+iind];
        double wi = d_u[2*NN+iind];

        double mi = d_m[iind];

        double dil = ZER;
        for (int64_t b = 0;b<NB;b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                int jind = iind + jadd[b];
                double uj = d_u[jind];
                double vj = d_u[NN+jind];
                double wj = d_u[2*NN+jind];
                double L0 = L0s[b];

                double A = dx2+uj-ui;
                double B = dy2+vj-vi;
                double C = dz2+wj-wi;
                double LN = sqrt(A*A + B*B + C*C);
                double eij = LN - L0;
                if (eij/L0 > ecrit){
                    SetBit(d_dmg, b*NN + iind);
                    printf("Bond broken");
                }else{
                    dil += eij*L*L*L;
                }

            }
        }
        d_dil[iind] = dil*3/mi;
    }
}

__global__ void calcForce(float *d_Sf, double *d_dil, double *d_u, bool *d_dmg, double *d_F, 
        double *d_vh, double *d_cd, double *d_cn, int *d_EBCi, double *d_k, double *d_W,
        double *d_Ft, double *d_m) {

    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF) + xl;
    float yi = L*(j+HLF) + yl;
    float zi = L*(k+HLF) + zl;

    if (Chi(xi,yi,zi)) {
        double ui = d_u[iind];
        double vi = d_u[NN+iind];
        double wi = d_u[2*NN+iind];
        double dili = d_dil[iind];
        double mi = d_m[iind];
        double ki = pow(d_k[iind],penal);

        double fx = ZER;
        double fy = ZER;
        double fz = ZER;

        double Wsm = ki*kappa*dili*dili/2;

        for (int64_t b = 0;b<NB;b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                int jind = iind + jadd[b];
                double uj = d_u[jind];
                double vj = d_u[NN+jind];
                double wj = d_u[2*NN+jind];
                double dilj = d_dil[jind];
                double L0 = L0s[b];
                double A = dx2+uj-ui;
                double B = dy2+vj-vi;
                double C = dz2+wj-wi;
                double LN = sqrt(A*A + B*B + C*C);
                double eij = LN - L0;
                double mj = d_m[jind];
                double kj = pow(d_k[jind],penal);
                // double fsm = (ki*dili + kj*dilj + (ki+kj)/2*eij/L0*fmlt);
                
                double tij = ki*(3*kappa*dili + am*(eij/L0 - dili/3))/mi; // ADD IN ki, kj!!
                double tji = kj*(3*kappa*dilj + am*(eij/L0 - dilj/3))/mj;
                double fsm = (tij + tji)*L*L*L;

                // double Wi = ki*(kappa*dili*dili/2/mi + am/mi/2*(eij/L0 - dili/3)*(eij - dili*L0/3));
                // double Wj = kj*(kappa*dilj*dilj/2/mj + am/mj/2*(eij/L0 - dilj/3)*(eij - dilj*L0/3));
                // Wsm += (Wi + Wj)/2;

                // double Wi = ki*(am*(eij/L0 - dili/3))/mi;
                // Wsm += Wi*Wi;


                // double Wi = ki*(1/L0)*(eij - dili*L0/3)*am/mi;
                // Wsm += Wi*Wi;

                double Wi = (ki*am/mi)*(kj*am/mj)*(1/L0)*(eij - dili*L0/3)*(eij - dili*L0/3);
                Wsm += Wi;


                fx += fsm*A/LN;
                fy += fsm*B/LN;
                fz += fsm*C/LN;
            }
        }

        int ebcx = d_EBCi[iind];
        int ebcy = d_EBCi[NN + iind];
        int ebcz = d_EBCi[2*NN + iind];
        double vhx = d_vh[iind];
        double vhy = d_vh[NN + iind];
        double vhz = d_vh[2*NN + iind];
        double pfx = d_F[iind];
        double pfy = d_F[NN + iind];
        double pfz = d_F[2*NN + iind];
        double cn = ZER;
        double cd = ZER;
        if(ebcx<0 && vhx != ZER){
            cn -= ui*ui*(fx - pfx)/(mvec*dt*vhx);
            cd += ui*ui;
        }
        if(ebcy<0 && vhy != ZER){
            cn -= vi*vi*(fy - pfy)/(mvec*dt*vhx);
            cd += vi*vi;
        }
        if(ebcz<0 && vhz != ZER){
            cn -= wi*wi*(fz - pfz)/(mvec*dt*vhx);
            cd += wi*wi;
        }
        d_cn[iind] = cn;
        d_cd[iind] = cd;
        
        d_F[iind] = fx;
        d_F[NN + iind] = fy;
        d_F[2*NN + iind] = fz;
        
        if(ebcx<0 && ebcy<0 && ebcz<0){
            d_Ft[iind] = sqrt(fx*fx + fy*fy + fz*fz);
            d_W[iind] = Wsm;
        }
    }
}

__global__ void calcDisplacement(double *d_c, double *d_u, double *d_vh, double *d_F, int *d_NBCi, float *d_NBC, int *d_EBCi, float *d_EBC){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;
    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    float xi = L*(i+HLF) + xl;
    float yi = L*(j+HLF) + yl;
    float zi = L*(k+HLF) + zl;
    double c = d_c[0];

    if (Chi(xi,yi,zi)) {
        double pfx = d_F[iind];
        double pfy = d_F[NN + iind];
        double pfz = d_F[2*NN + iind];
        double bfx = ZER; 
        double bfy = ZER; 
        double bfz = ZER;
        int nbci = d_NBCi[iind];
        if(nbci>=0){
            bfx = d_NBC[3*nbci];
            bfy = d_NBC[3*nbci + 1];
            bfz = d_NBC[3*nbci + 2];
        }
        double vhox = d_vh[iind];
        double vhoy = d_vh[NN + iind];
        double vhoz = d_vh[2*NN + iind];
        double ui = d_u[iind];
        double vi = d_u[NN + iind];
        double wi = d_u[2*NN + iind];

        double vhx; double vhy; double vhz;

        if (tt==0){
            vhx = dt/mvec * (pfx + bfx) / TWO;
            vhy = dt/mvec * (pfy + bfy) / TWO;
            vhz = dt/mvec * (pfz + bfz) / TWO;
        } else {
            vhx = ((TWO - c*dt)*vhox + TWO*dt/mvec*(pfx + bfx))/(TWO + c*dt);
            vhy = ((TWO - c*dt)*vhoy + TWO*dt/mvec*(pfy + bfy))/(TWO + c*dt);
            vhz = ((TWO - c*dt)*vhoz + TWO*dt/mvec*(pfz + bfz))/(TWO + c*dt);
        }

        
        int ebcx = d_EBCi[iind];
        int ebcy = d_EBCi[NN + iind];
        int ebcz = d_EBCi[2*NN + iind];
        if(ebcx<0){
            d_u[iind] = ui + dt*vhx;
        }else{
            d_u[iind] = d_EBC[ebcx]*min(ONE, tt/ntau);
        }
        if(ebcy<0){
            d_u[NN+iind] = vi + dt*vhy;
        }else{
            d_u[NN+iind] = d_EBC[ebcy]*min(ONE, tt/ntau);
        }
        if(ebcz<0){
            d_u[2*NN + iind] = wi + dt*vhz;
        }else{
            d_u[2*NN + iind] = d_EBC[ebcz]*min(ONE, tt/ntau);
        }
        d_vh[iind] = vhx;
        d_vh[NN + iind] = vhy;
        d_vh[2*NN + iind] = vhz;
    }
}

__global__ void calcKbar(float *d_Sf, double *d_Wt, double *d_RM, double *d_W, bool *d_dmg, double *d_kbar){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF) + xl;
    float yi = L*(j+HLF) + yl;
    float zi = L*(k+HLF) + zl;

    double Wt = d_Wt[0];
    double RM = d_RM[0];

    if (Chi(xi,yi,zi)) {
        double kopti = d_W[iind] *NN * RM / Wt;
        double nsm = kopti*hrad;
        double dsm = hrad;
        for (int64_t b = 0; b<NB; b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                int jind = iind + jadd[b];
                double psi = max(ZER, hrad - L0s[b]);
                nsm += psi * d_W[jind] * NN * RM / Wt;
                dsm += psi;
            }
        }
        d_kbar[iind] = max(0.00001, min(ONE, d_kbar[iind] + nsm / dsm));
    }
}

__global__ void updateK(double *d_k, double *d_kbar, int *d_NBCi, int *d_EBCi){
    int iind = blockIdx.x * blockDim.x + threadIdx.x;

    if(iind<NN){
        if(d_NBCi[iind]<0 && d_EBCi[iind]<0 && d_EBCi[NN + iind]<0 && d_EBCi[2*NN + iind]<0){
            d_k[iind] = alpha*d_k[iind] + (1 - alpha) * d_kbar[iind];
            d_kbar[iind] = ZER;
        }
    }
}


