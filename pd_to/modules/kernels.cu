#include "hip/hip_runtime.h"
#include <math.h>
#include <inttypes.h>


__device__ __constant__ float L = Lr;
__device__ __constant__ int NX = NXr;
__device__ __constant__ int NY = NYr;
__device__ __constant__ int NZ = NZr;
__device__ __constant__ int NB = NBr;
__device__ __constant__ int NN = NNr;
__device__ __constant__ double L0s[];
__device__ __constant__ int jadd[];
__device__ __constant__ double dt = dtr;
__device__ __constant__ double ntau = ntaur;
__device__ __constant__ double rho = rhor;
__device__ __constant__ double ecrit = ecritr;

__device__ __constant__ double dlmlt = dlmltr;
__device__ __constant__ double fmlt = fmltr;
__device__ __constant__ double mvec = mvecr;

__device__ __constant__ float ZERf = 0;
__device__ __constant__ float ONEf = 1;
__device__ __constant__ float HLF = 0.5;
__device__ __constant__ float TRE = 3;

__device__ __constant__ double ZER = 0;
__device__ __constant__ double ONE = 1;
__device__ __constant__ double TWO = 2;

__device__ int tt = 0;

__device__ __shared__ float sh_Sf[SHr]; // 4*NB + 1

__device__ bool Chi(float x, float y, float z){
    return (x>=ZERf) && (x<=ONEf) && (y>=ZERf) && (y<=ONEf) && (z>=ZERf) && (z<=ONEf);
}

__device__ bool TestBit(bool A[],  int64_t k ){
    return ( (A[k/8] & (1 << (k%8) )) != 0 ) ;     
}

__device__ void  SetBit(bool A[],  int64_t k ){
    A[k/8] |= 1 << (k%8);
}

__global__ void calcDilation(float *d_Sf, double *d_u, double *d_dil, bool *d_dmg){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(iind==0){
        tt += 1;
    }

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF);
    float yi = L*(j+HLF);
    float zi = L*(k+HLF);

    if (Chi(xi,yi,zi)) {
        double ui = d_u[iind];
        double vi = d_u[NN+iind];
        double wi = d_u[2*NN+iind];

        double dil = ZER;
        for (int64_t b = 0;b<NB;b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                int jind = iind + jadd[b];
                double uj = d_u[jind];
                double vj = d_u[NN+jind];
                double wj = d_u[2*NN+jind];
                double L0 = L0s[b];
                double A = dx2+uj-ui;
                double B = dy2+vj-vi;
                double C = dz2+wj-wi;
                double LN = sqrt(A*A + B*B + C*C);
                double eij = LN - L0;
                if (eij/L0 > ecrit){
                    SetBit(d_dmg, b*NN + iind);
                    printf("Bond broke");
                }else{
                    dil += eij/L0;
                }
            }
        }
        d_dil[iind] = dil*dlmlt;
    }
}

__global__ void calcForce(float *d_Sf, double *d_dil, double *d_u, bool *d_dmg, double *d_F, double *d_vh, double *d_cd, double *d_cn, int *d_EBCi) {
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;

    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    if(tix<NB){
        sh_Sf[tix] = d_Sf[tix];
        sh_Sf[NB+tix] = d_Sf[NB+tix];
        sh_Sf[2*NB+tix] = d_Sf[2*NB+tix];
    }
    __syncthreads();

    float xi = L*(i+HLF);
    float yi = L*(j+HLF);
    float zi = L*(k+HLF);

    if (Chi(xi,yi,zi)) {
        double ui = d_u[iind];
        double vi = d_u[NN+iind];
        double wi = d_u[2*NN+iind];
        double dili = d_dil[iind];

        double fx = ZER;
        double fy = ZER;
        double fz = ZER;

        for (int64_t b = 0;b<NB;b++){
            float dx2 = sh_Sf[b];
            float dy2 = sh_Sf[NB+b];
            float dz2 = sh_Sf[2*NB+b];
            if (Chi(xi+dx2, yi+dy2, zi+dz2) && !TestBit(d_dmg, b*NN + iind)) {
                int jind = iind + jadd[b];
                double uj = d_u[jind];
                double vj = d_u[NN+jind];
                double wj = d_u[2*NN+jind];
                double dilj = d_dil[jind];
                double L0 = L0s[b];
                double A = dx2+uj-ui;
                double B = dy2+vj-vi;
                double C = dz2+wj-wi;
                double LN = sqrt(A*A + B*B + C*C);
                double eij = LN - L0;
                double fsm = dili + dilj + eij/L0*fmlt;
                double dln = fsm/LN;
                fx += dln*A;
                fy += dln*B;
                fz += dln*C;
            }
        }

        int ebcx = d_EBCi[iind];
        int ebcy = d_EBCi[NN + iind];
        int ebcz = d_EBCi[2*NN + iind];
        double vhx = d_vh[iind];
        double vhy = d_vh[NN + iind];
        double vhz = d_vh[2*NN + iind];
        double pfx = d_F[iind];
        double pfy = d_F[NN + iind];
        double pfz = d_F[2*NN + iind];
        double cn = ZER;
        double cd = ZER;
        if(ebcx<0 && vhx != ZER){
            cn -= ui*ui*(fx - pfx)/(mvec*dt*vhx);
            cd += ui*ui;
        }
        if(ebcy<0 && vhy != ZER){
            cn -= vi*vi*(fy - pfy)/(mvec*dt*vhx);
            cd += vi*vi;
        }
        if(ebcz<0 && vhz != ZER){
            cn -= wi*wi*(fz - pfz)/(mvec*dt*vhx);
            cd += wi*wi;
        }
        d_cn[iind] = cn;
        d_cd[iind] = cd;
        
        d_F[iind] = fx;
        d_F[NN + iind] = fy;
        d_F[2*NN + iind] = fz;
    }
}

__global__ void calcDisplacement(double *d_c, double *d_u, double *d_vh, double *d_F, int *d_NBCi, float *d_NBC, int *d_EBCi, float *d_EBC){
    int tix = threadIdx.x;
    int iind = blockIdx.x * blockDim.x + tix;
    int k = iind/(NX*NY);
    int j = iind%(NX*NY)/NX;
    int i = iind%NX;

    float xi = L*(i+HLF);
    float yi = L*(j+HLF);
    float zi = L*(k+HLF);
    double c = d_c[0];

    if (Chi(xi,yi,zi)) {
        double pfx = d_F[iind];
        double pfy = d_F[NN + iind];
        double pfz = d_F[2*NN + iind];
        double bfx = ZER; 
        double bfy = ZER; 
        double bfz = ZER;
        int nbci = d_NBCi[iind];
        if(nbci>=0){
            bfx = d_NBC[3*nbci];
            bfy = d_NBC[3*nbci + 1];
            bfz = d_NBC[3*nbci + 2];
        }
        double vhox = d_vh[iind];
        double vhoy = d_vh[NN + iind];
        double vhoz = d_vh[2*NN + iind];
        double ui = d_u[iind];
        double vi = d_u[NN + iind];
        double wi = d_u[2*NN + iind];

        double vhx; double vhy; double vhz;

        if (tt==0){
            vhx = dt/mvec * (pfx + bfx) / TWO;
            vhy = dt/mvec * (pfy + bfy) / TWO;
            vhz = dt/mvec * (pfz + bfz) / TWO;
        } else {
            vhx = ((TWO - c*dt)*vhox + TWO*dt/mvec*(pfx + bfx))/(TWO + c*dt);
            vhy = ((TWO - c*dt)*vhoy + TWO*dt/mvec*(pfy + bfy))/(TWO + c*dt);
            vhz = ((TWO - c*dt)*vhoz + TWO*dt/mvec*(pfz + bfz))/(TWO + c*dt);
        }

        
        int ebcx = d_EBCi[iind];
        int ebcy = d_EBCi[NN + iind];
        int ebcz = d_EBCi[2*NN + iind];
        if(ebcx<0){
            d_u[iind] = ui + dt*vhx;
        }else{
            d_u[iind] = d_EBC[ebcx]*min(ONE, tt/ntau);
        }
        if(ebcy<0){
            d_u[NN+iind] = vi + dt*vhy;
        }else{
            d_u[NN+iind] = d_EBC[ebcy]*min(ONE, tt/ntau);
        }
        if(ebcz<0){
            d_u[2*NN + iind] = wi + dt*vhz;
        }else{
            d_u[2*NN + iind] = d_EBC[ebcz]*min(ONE, tt/ntau);
        }
        d_vh[iind] = vhx;
        d_vh[NN + iind] = vhy;
        d_vh[2*NN + iind] = vhz;
    }
}